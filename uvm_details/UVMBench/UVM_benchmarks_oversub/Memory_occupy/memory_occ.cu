#include <iostream>
#include <math.h>
#include <stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>

#include <unistd.h>
#include <assert.h>

// #define GB 8
// #define MB 1
// #define KB 1

int main(int argc, char *argv[])
{
    unsigned GB,MB,KB;
    char *endptr;

    fprintf(stdout, "Input %d argument\n",argc);
    assert(argc == 3);
    // float size_char_format = atoi(argv[1]);
    float size = strtof(argv[1], &endptr);
    if (*endptr != '\0')
        printf("%s is not convertible to float\n", argv[1]);
    else
        printf("size is  %f\n", size ); //0.000000    
    int over_sub_char_format = atoi(argv[2]);
    // float size = (float) size_char_format;
    unsigned over_sub = (unsigned) over_sub_char_format;

    if (over_sub == 1)
    {
        // printf("size is %f\n", size);
        fprintf(stdout, "It is 110%% oversubscription mode\n");
        float occupy_mem = 11  - size   +  size * 1.0/10.0 -0.1 ;
        GB = (unsigned) occupy_mem;
        float temp_MB = (occupy_mem -floor(occupy_mem)) *1024.0;
        MB = (unsigned) temp_MB;
        float temp_KB = (temp_MB - floor(temp_MB)) *1024.0;
        KB = (unsigned) temp_KB;
    }
    else if (over_sub == 2)
    {
        printf("size is %f\n", size);
        fprintf(stdout, "It is 125%% oversubscription mode\n");
        float occupy_mem = 11  - size   +  size * 1.0/4.0 - 0.1  ; 
        GB = (unsigned) occupy_mem;
        float temp_MB = (occupy_mem -floor(occupy_mem)) *1024.0;
        MB = (unsigned) temp_MB;
        float temp_KB = (temp_MB - floor(temp_MB)) *1024.0;
        KB = (unsigned) temp_KB;
    }else{
        assert(0);
    }
    
    if (GB == 0)
    {
        assert(0);
    }
    if(MB == 0)
    {
        assert(0);
    }
    if(KB == 0)
    {
        assert(0);
    }
    
    fprintf(stdout, "the calculated size is %u GB, %u MB,%u KB \n",GB,MB,KB);
    

    float *GB_ptr;
    float *MB_ptr;
    float *KB_ptr;
    
    hipMalloc((void **) &GB_ptr, sizeof(float) * 1024*1024*256 * GB );
    hipMalloc((void **) &MB_ptr, sizeof(float) * 1024*256 * MB);
    hipMalloc((void **) &KB_ptr, sizeof(float) * 256 * KB);
    
    sleep(900);
    sleep(900);
    sleep(900);
    sleep(900);
    sleep(900);
    

    hipFree(GB_ptr);
    hipFree(MB_ptr);
    hipFree(KB_ptr);
    return 0;
}